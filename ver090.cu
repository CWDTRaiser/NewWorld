#include "hip/hip_runtime.h"
//PND
#define X 1
//RN
#define Y 10
//membrane potential increase per neurotransmitter
#define Z 100
//CD
#define C 10
//MPD
#define D 5
#include <hip/hip_runtime.h>
#include "menu.h"
using namespace std;

map<int, neuronTag> Tag;
/*
void check_preNeuron(){
    
}
*/
void gen_neuron(int i){
    neuronTag *collect = NULL;
    collect = new neuronTag [i];
    
    for(int id = 1; id < i + 1; id++){
        collect[id].set_ID(id);
        collect[id].gen_NT();
        collect[id].NT -> NeuronSave();
        Tag.insert(pair<int, neuronTag>(id,collect[id]));
    }

}

string Nread_word(int number){
    string temp_buffer = "";
    for(int i = 0; i < number; i++){
        test >> temp_buffer;
    }
    return temp_buffer;
    //number is the thread of word
}

int get_neuron_check(){
    int tempN = 1;
    int neuronNumber = 0;
    for(int i = 0; i < 1; i++){
        string buffer;
        char tempput[15] = "";
        sprintf(datanumber, "%d", tempN);
        strcat(tempput, output);
        strcat(tempput, datanumber);
        strcat(tempput, format);
        test.open(tempput);
        if(test.is_open()){
            i = i - 1;
            neuronNumber = neuronNumber + 1;
            
            //cout << "check" ;
        }else{
            //cout << "note" << tempN << " no such file" << endl;
        }
        test.close();
        tempN = tempN + 1;
    }
    return neuronNumber;
}

void get_neuron(){
    int neuronNumber = get_neuron_check();
    neuronTag *collect = NULL;
    collect = new neuronTag [neuronNumber];
    //temp
    for(int i = 1/*need to change to log*/; i < neuronNumber + 1; i++){
        collect[i].NT = new neuron(i); 
        Tag.insert(pair<int, neuronTag>(i,collect[i]));
        char tempput[15] = "";
        sprintf(datanumber, "%d", i);
        strcat(tempput, output);
        strcat(tempput, datanumber);
        strcat(tempput, format);
        
        test.open(tempput, ios::in);
        Tag[i].NT -> name = atoi(Nread_word(3).c_str());
        Tag[i].NT -> CD = atoi(Nread_word(4).c_str());
        Tag[i].NT -> TH = atoi(Nread_word(3).c_str());
        Tag[i].NT -> NS = atoi(Nread_word(4).c_str());
        Tag[i].NT -> PND = atoi(Nread_word(5).c_str());
        Tag[i].NT -> MPD = atoi(Nread_word(5).c_str());
        Tag[i].NT -> RN = atoi(Nread_word(5).c_str());
        
        test.close();
    }
} 


int main(){
    srand(time(NULL));
    string temp_input;
    
    
    //initialization of neuron
    double START, END;
    START = clock();
    cout << get_neuron_check();
    if(get_neuron_check() != 0){
        cout << "!=0" << endl;
        get_neuron();
    }else{
        cout << "==0" << endl;
        gen_neuron(8);    
    }
    END = clock();
    
    //check log and get log
    test.open("log");
    if(test.is_open()){
    
    }else{
        test.open("log", ios::out);
        test << 0 << endl;
        test.close();
    }
    ifstream read("log", ios::in);
    read >> temp_input;
    read.close();
    int pastlog = atoi(temp_input.c_str());
    temp_input = "";
    /*
    cout << pastlog << endl;
    test.open("log", ios::out);
    test << "HAHAHA" << endl;
    test << pastlog << endl;
    test.close();
    */

    for(int i = pastlog; i < pastlog + 100; i++){
        for(int j = 1; j < 9; j++){
            if(Tag[j].NT -> at == i){
                Tag[j].NT -> NeuronExe();
                if(j == 8){
                    Tag[1].NT -> PreN = Tag[1].NT -> PreN + (Tag[j].NT -> OT) * (Tag[j].NT -> RN );
                }else{
                    Tag[j+1].NT -> PreN = Tag[j+1].NT -> PreN + (Tag[j].NT -> OT) * (Tag[j].NT -> RN );
                }
            }

            Tag[j].NT -> NeuronNote(i);
            Tag[j].NT -> Neurontimepass();
        }
    }
    /*
    test.open("log", ios::out | ios::app);
    test << pastlog << endl;
    test.close();
    */
    cout << (double)clock()/CLOCKS_PER_SEC << " s" << endl;
    cout << (END-START)/1000000 << " s" << endl;
    return 0;
}
